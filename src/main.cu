#include "hip/hip_runtime.h"
// src/main.cu
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <filesystem>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>

#ifndef NUM_STREAMS
#define NUM_STREAMS 4
#endif

#define CUDA_CHECK(x) do { \
  hipError_t err = (x); \
  if (err != hipSuccess) { \
    std::cerr << "CUDA error " << __FILE__ << ":" << __LINE__ << " - " \
              << hipGetErrorString(err) << std::endl; \
    std::exit(EXIT_FAILURE); \
  } \
} while (0)

struct DeviceBuf {
    unsigned char* d_in  = nullptr;
    unsigned char* d_out = nullptr;
    size_t bytes = 0;
    hipStream_t stream = nullptr;
};

__global__ void stylize_kernel(
    const unsigned char* __restrict__ in,
    unsigned char* __restrict__ out,
    int num_pixels, float alpha, float beta)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_pixels) return;
    int base = idx * 3; // BGR
    // Simple brightness/contrast-ish per-pixel op:
    float b = in[base + 0] * alpha + beta;
    float g = in[base + 1] * alpha + beta;
    float r = in[base + 2] * alpha + beta;
    // mild channel remap for visible change
    out[base + 0] = (unsigned char)max(0.f, min(255.f, 0.7f * b + 0.3f * g));
    out[base + 1] = (unsigned char)max(0.f, min(255.f, 0.7f * g + 0.3f * r));
    out[base + 2] = (unsigned char)max(0.f, min(255.f, 0.7f * r + 0.3f * b));
}

static void ensure_dir(const std::string& p) {
    std::filesystem::create_directories(p);
}

int main(int argc, char** argv) {
    if (argc < 4) {
        std::cout << "Usage: " << argv[0]
                  << " <input_dir> <output_dir> <alpha> [beta] [num_streams]\n"
                  << "Example: " << argv[0] << " data/input output 1.2 20 4\n";
        return 1;
    }
    std::string input_dir = argv[1];
    std::string output_dir = argv[2];
    float alpha = std::stof(argv[3]);
    float beta  = (argc >= 5) ? std::stof(argv[4]) : 0.0f;
    int n_streams = (argc >= 6) ? std::stoi(argv[5]) : NUM_STREAMS;

    ensure_dir(output_dir);

    // Collect images
    std::vector<std::string> paths;
    for (auto& e : std::filesystem::directory_iterator(input_dir)) {
        if (!e.is_regular_file()) continue;
        auto ext = e.path().extension().string();
        std::transform(ext.begin(), ext.end(), ext.begin(), ::tolower);
        if (ext == ".jpg" || ext == ".png" || ext == ".jpeg" || ext == ".bmp" || ext==".tif" || ext==".tiff")
            paths.push_back(e.path().string());
    }
    if (paths.empty()) {
        std::cerr << "No images found in " << input_dir << "\n";
        return 1;
    }

    std::cout << "Found " << paths.size() << " images\n";

    // Create streams and per-stream device buffers (ping-pong over streams)
    n_streams = std::max(1, n_streams);
    std::vector<DeviceBuf> dev(n_streams);

    for (int i = 0; i < n_streams; ++i) {
        CUDA_CHECK(hipStreamCreate(&dev[i].stream));
    }

    std::ofstream log_csv(output_dir + "/timings.csv");
    log_csv << "image,rows,cols,bytes,ms_h2d,ms_kernel,ms_d2h,total_ms\n";

    auto t0_all = std::chrono::high_resolution_clock::now();

    for (size_t i = 0; i < paths.size(); ++i) {
        const int sid = static_cast<int>(i % n_streams);

        cv::Mat img = cv::imread(paths[i], cv::IMREAD_COLOR);
        if (img.empty()) {
            std::cerr << "Failed to read: " << paths[i] << "\n";
            continue;
        }
        if (!img.isContinuous()) img = img.clone();

        const int rows = img.rows, cols = img.cols;
        const int num_pixels = rows * cols;
        const size_t bytes = static_cast<size_t>(num_pixels) * 3;

        // (Re)alloc per stream buffers if needed
        if (bytes != dev[sid].bytes) {
            if (dev[sid].d_in)  CUDA_CHECK(hipFree(dev[sid].d_in));
            if (dev[sid].d_out) CUDA_CHECK(hipFree(dev[sid].d_out));
            CUDA_CHECK(hipMalloc(&dev[sid].d_in,  bytes));
            CUDA_CHECK(hipMalloc(&dev[sid].d_out, bytes));
            dev[sid].bytes = bytes;
        }

        cv::Mat out(rows, cols, CV_8UC3);
        hipEvent_t e_h2d_start, e_h2d_end, e_k_start, e_k_end, e_d2h_start, e_d2h_end;
        CUDA_CHECK(hipEventCreate(&e_h2d_start));
        CUDA_CHECK(hipEventCreate(&e_h2d_end));
        CUDA_CHECK(hipEventCreate(&e_k_start));
        CUDA_CHECK(hipEventCreate(&e_k_end));
        CUDA_CHECK(hipEventCreate(&e_d2h_start));
        CUDA_CHECK(hipEventCreate(&e_d2h_end));

        CUDA_CHECK(hipEventRecord(e_h2d_start, dev[sid].stream));
        CUDA_CHECK(hipMemcpyAsync(dev[sid].d_in, img.ptr<unsigned char>(), bytes,
                                   hipMemcpyHostToDevice, dev[sid].stream));
        CUDA_CHECK(hipEventRecord(e_h2d_end, dev[sid].stream));

        int threads = 256;
        int blocks = (num_pixels + threads - 1) / threads;
        CUDA_CHECK(hipEventRecord(e_k_start, dev[sid].stream));
        stylize_kernel<<<blocks, threads, 0, dev[sid].stream>>>(
            dev[sid].d_in, dev[sid].d_out, num_pixels, alpha, beta);
        CUDA_CHECK(hipEventRecord(e_k_end, dev[sid].stream));

        CUDA_CHECK(hipEventRecord(e_d2h_start, dev[sid].stream));
        CUDA_CHECK(hipMemcpyAsync(out.ptr<unsigned char>(), dev[sid].d_out, bytes,
                                   hipMemcpyDeviceToHost, dev[sid].stream));
        CUDA_CHECK(hipEventRecord(e_d2h_end, dev[sid].stream));

        CUDA_CHECK(hipStreamSynchronize(dev[sid].stream));

        float ms_h2d=0, ms_k=0, ms_d2h=0;
        CUDA_CHECK(hipEventElapsedTime(&ms_h2d, e_h2d_start, e_h2d_end));
        CUDA_CHECK(hipEventElapsedTime(&ms_k,   e_k_start,   e_k_end));
        CUDA_CHECK(hipEventElapsedTime(&ms_d2h, e_d2h_start, e_d2h_end));

        float total_ms = ms_h2d + ms_k + ms_d2h;

        std::string fname = std::filesystem::path(paths[i]).filename().string();
        std::string out_path = output_dir + "/proc_" + fname;
        cv::imwrite(out_path, out);

        log_csv << fname << "," << rows << "," << cols << "," << bytes
                << "," << ms_h2d << "," << ms_k << "," << ms_d2h << "," << total_ms << "\n";

        CUDA_CHECK(hipEventDestroy(e_h2d_start));
        CUDA_CHECK(hipEventDestroy(e_h2d_end));
        CUDA_CHECK(hipEventDestroy(e_k_start));
        CUDA_CHECK(hipEventDestroy(e_k_end));
        CUDA_CHECK(hipEventDestroy(e_d2h_start));
        CUDA_CHECK(hipEventDestroy(e_d2h_end));

        if ((i+1) % 10 == 0) {
            std::cout << "Processed " << (i+1) << "/" << paths.size() << " images\r";
        }
    }

    for (int i = 0; i < n_streams; ++i) {
        if (dev[i].d_in)  hipFree(dev[i].d_in);
        if (dev[i].d_out) hipFree(dev[i].d_out);
        hipStreamDestroy(dev[i].stream);
    }

    auto t1_all = std::chrono::high_resolution_clock::now();
    double total_s = std::chrono::duration<double>(t1_all - t0_all).count();
    std::cout << "\nDone. Total wall time: " << total_s << " s\n";
    log_csv.close();

    return 0;
}
